/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void staticReverse(int *d, int n) {
  __shared__ int s[64];
  int t = threadIdx.x;
  int tr = n - t - 1;
  s[t] = d[t];
  __syncthreads();
  d[t] = s[tr];
}

__global__ void dynamicReverse(int *d, int n) {
  extern __shared__ int s[];
  int t = threadIdx.x;
  int tr = n - t - 1;
  s[t] = d[t];
  __syncthreads();
  d[t] = s[tr];
}

int main(void) {
  const int n = 64;
  int a[n], r[n], d[n];

  for (int i = 0; i < n; i++) {
    a[i] = i;
    r[i] = n - i - 1;
    d[i] = 0;
  }

  int *d_d;
  hipMalloc(&d_d, n * sizeof(int));

  // run version with static shared memory
  hipMemcpy(d_d, a, n * sizeof(int), hipMemcpyHostToDevice);
  staticReverse<<<1, n>>>(d_d, n);
  hipMemcpy(d, d_d, n * sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < n; i++)
    if (d[i] != r[i])
      printf("Error: d[%d]!=r[%d] (%d, %d)\n", i, i, d[i], r[i]);

  // run dynamic shared memory version
  hipMemcpy(d_d, a, n * sizeof(int), hipMemcpyHostToDevice);
  dynamicReverse<<<1, n, n * sizeof(int)>>>(d_d, n);
  hipMemcpy(d, d_d, n * sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < n; i++)
    if (d[i] != r[i])
      printf("Error: d[%d]!=r[%d] (%d, %d)\n", i, i, d[i], r[i]);
}
